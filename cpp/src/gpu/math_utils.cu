#include <hip/hip_runtime.h>

// Simple GPU kernel
__global__ void add_kernel(const int* a, const int* b, int* c) {
    *c = *a + *b;
}

extern "C" int add_gpu_impl(int a, int b) {
    int *d_a, *d_b, *d_c;
    int result;

    hipMalloc((void**)&d_a, sizeof(int));
    hipMalloc((void**)&d_b, sizeof(int));
    hipMalloc((void**)&d_c, sizeof(int));

    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

    add_kernel<<<1, 1>>>(d_a, d_b, d_c);
    hipMemcpy(&result, d_c, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return result;
}
